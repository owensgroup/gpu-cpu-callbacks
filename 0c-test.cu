#define CUDA_SAFE_CALL(x)                                                                                         \
{                                                                                                                 \
  printf("%s.%s.%d: %s\n", __FILE__, __FUNCTION__, __LINE__, #x); fflush(stdout);                                 \
  hipError_t error = (x);                                                                                        \
  if (error != hipSuccess)                                                                                       \
  {                                                                                                               \
    fprintf(stderr, "%s.%s.%d: 0x%x (%s)\n", __FILE__, __FUNCTION__, __LINE__, error, hipGetErrorString(error)); \
    hipGetLastError();                                                                                           \
    exit(1);                                                                                                      \
  }                                                                                                               \
}                                                                                                                 \


#include <hip/hip_runtime.h>
#include <cstdio>

int main(int argc, char ** argv)
{
  dim3 gs(5, 5, 1);
  dim3 bs(4, 4, 4);
  int * cpuMem, * gpuMem;

  CUDA_SAFE_CALL(hipSetDeviceFlags(hipDeviceMapHost));
  CUDA_SAFE_CALL(hipSetDevice(0));
  CUDA_SAFE_CALL(hipHostAlloc(reinterpret_cast<void ** >(&cpuMem), sizeof(int) * 25 * 64, hipHostMallocMapped));
  CUDA_SAFE_CALL(hipHostGetDevicePointer(reinterpret_cast<void ** >(&gpuMem), cpuMem, 0));

  fprintf(stderr, "cpuMem gpuMem { %p %p }\n", cpuMem, gpuMem); fflush(stderr);

  return 0;
}
