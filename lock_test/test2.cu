#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#define CUDA_SAFE_CALL(x) { hipError_t error = (x); if (error != hipSuccess) { fprintf(stderr, "%s.%s.%d: 0x%x (%s)\n", __FILE__, __FUNCTION__, __LINE__, error, hipGetErrorString(error)); hipGetLastError(); exit(1); } }

__device__ unsigned int lock;
__device__ volatile unsigned int counter;

__global__ void kernel()
{
  if (threadIdx.x % 32 == 0)
  {
    while (atomicExch(&lock, 1) == 1) { }
    ++counter;
    atomicExch(&lock, 0);
  }
}

int main(int argc, char ** argv)
{
  unsigned int * gpuLock, * gpuCounter;
  int cpuLock, cpuCounter;
  CUDA_SAFE_CALL(hipSetDevice(0));
  CUDA_SAFE_CALL(hipGetSymbolAddress(reinterpret_cast<void ** >(&gpuLock), lock));
  CUDA_SAFE_CALL(hipGetSymbolAddress(reinterpret_cast<void ** >(&gpuCounter), counter));

  dim3 gs(1, 1, 1), bs(64, 1, 1);
  CUDA_SAFE_CALL(hipMemset(gpuLock,    0, sizeof(unsigned int)));
  CUDA_SAFE_CALL(hipMemset(gpuCounter, 0, sizeof(unsigned int)));
  kernel<<<gs, bs>>>();
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(hipMemcpy(&cpuLock,     gpuLock,    sizeof(unsigned int), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(&cpuCounter,  gpuCounter, sizeof(unsigned int), hipMemcpyDeviceToHost));

  return 0;
}
